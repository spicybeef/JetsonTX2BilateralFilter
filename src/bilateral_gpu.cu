#include "hip/hip_runtime.h"
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "bilateral_gpu.h"

const int BLOCKDIM = 16;

/**
 * @brief      Calculates the Euclidean distance between two points (x0, y0) and
 *             (x1, y1)
 *
 * @param[in]  x0    The x0 coordinate
 * @param[in]  y0    The y0 coordinate
 * @param[in]  x1    The x1 coordinate
 * @param[in]  y1    The y1 coordinate
 *
 * @return     The distance between the two points
 */
__device__ float distance(int x0, int y0, int x1, int y1)
{
    return static_cast<float>(sqrtf( (x0 - x1) * (x0 - x1) + (y0 - y1) * (y0 - y1) ));
}

/**
 * @brief      Calculates the one-dimensional Gaussian function for a given
 *             point x
 *
 * @param[in]  x      The point on the distribution to return a value from
 * @param[in]  mu     The mean value
 * @param[in]  sigma  The standard deviation of the distribution
 *
 * @return     The value of the 1D Gaussian function at point x with mean mu and
 *             standard deviation sigma
 */
__device__ float gaussian(float x, float mu, float sigma)
{
    return static_cast<float>(exp(-((x - mu) * (x - mu))/(2 * sigma * sigma)) / (2 * M_PI * sigma * sigma));
}

/**
 * @brief      A naive implementation of the bilateral filter
 *
 * @param      inputImage   The input float array
 * @param      outputImage  The output float array
 * @param[in]  rows         The number of rows in the image
 * @param[in]  cols         The number of columns in the image
 * @param[in]  window       The window to use in the filter
 * @param[in]  sigmaD       The distance parameter
 * @param[in]  sigmaR       The intensity parameter
 */
__global__ void bilateralGpuKernel(
    float* inputImage,
    float* outputImage,
    int rows, int cols,
    uint32_t window,
    float sigmaD,
    float sigmaR)
{
    float filteredPixel;
    float wP, gR, gD;
    int neighborCol;
    int neighborRow;

    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col >= cols || row >= rows)
    {
        return;
    }

    filteredPixel = 0;
    wP = 0;

    for (int windowCol = 0; windowCol < window; windowCol++)
    {
        for (int windowRow = 0; windowRow < window; windowRow++)
        {
            neighborCol = col - (window / 2) - windowCol;
            neighborRow = row - (window / 2) - windowRow;

            // Prevent us indexing into regions that don't exist
            if (neighborCol < 0)
            {
                neighborCol = 0;
            }
            if (neighborRow < 0)
            {
                neighborRow = 0;
            }

            // Intensity factor
            gR = gaussian(inputImage[neighborCol + neighborRow * cols] - inputImage[col + row * cols], 0.0, sigmaR);
            // Distance factor
            gD = gaussian(distance(col, row, neighborCol, neighborRow), 0.0, sigmaD);

            filteredPixel += inputImage[neighborCol + neighborRow * cols] * (gR * gD);

            wP += (gR * gD);
        }
    }
    outputImage[col + row * cols] = filteredPixel / wP;
}

void bilateralNaiveGpu(
    float* inputImage,
    float* outputImage,
    int rows, int cols,
    uint32_t window,
    float sigmaD,
    float sigmaR)
{
    float* gpuInput;
    float* gpuOutput;
    hipError_t cudaStatus; 
    
    cudaStatus = hipMalloc<float>(&gpuInput, rows * cols * sizeof(float));
    checkCudaErrors(cudaStatus);    
    cudaStatus = hipMalloc<float>(&gpuOutput, rows * cols * sizeof(float));
    checkCudaErrors(cudaStatus);

    cudaStatus = hipMemcpy(gpuInput, inputImage, rows * cols * sizeof(float), hipMemcpyHostToDevice);
    checkCudaErrors(cudaStatus);    
    
    const dim3 block(BLOCKDIM, BLOCKDIM);
    const dim3 grid((cols / BLOCKDIM) + (cols % BLOCKDIM), (rows / BLOCKDIM) + (rows % BLOCKDIM));

    // printf("BlockDimensions x=%d, y=%d, z=%d\n", block.x, block.y, block.z);
    // printf("GridDimensions x=%d, y=%d, z=%d\n", grid.x, grid.y, grid.z);

    bilateralGpuKernel<<<grid,block>>>(gpuInput, gpuOutput, rows, cols, window, sigmaD, sigmaR);

    cudaStatus = hipDeviceSynchronize();
    checkCudaErrors(cudaStatus);    

    cudaStatus = hipMemcpy(outputImage, gpuOutput, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
    checkCudaErrors(cudaStatus);    

    cudaStatus = hipFree(gpuInput);
    checkCudaErrors(cudaStatus);    
    cudaStatus = hipFree(gpuOutput);
    checkCudaErrors(cudaStatus);    
}
